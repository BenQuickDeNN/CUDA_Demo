#include "cuda_getGPUInfo.h"

#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

//void getGPUInfo();

/* ͳ��GPU��Ϣ */
void getGPUInfo()
{
	printf("----------------ͳ��GPU��Ϣ----------------\r\n");
	int dev = 0;				// �豸
	hipDeviceProp_t devProp;		// �豸����
	hipGetDeviceProperties(&devProp, dev);
	printf("ʹ��GPU device %d: %s\r\n", dev, devProp.name);
	printf("SM������: %d\r\n", devProp.multiProcessorCount);
	printf("ÿ���߳̿�Ĺ����ڴ��С: %f KB\r\n", devProp.sharedMemPerBlock / 1024.0);
	printf("ÿ���߳̿������߳���: %d\r\n", devProp.maxThreadsPerBlock);
	printf("ÿ��SM������߳���: %d\r\n", devProp.maxThreadsPerMultiProcessor);
	printf("�߳�����С: %d\r\n", devProp.warpSize);
	printf("ÿ��SM������߳�����: %d\r\n", devProp.maxThreadsPerMultiProcessor / devProp.warpSize);
	//printf("�������ߴ�: %ld\r\n", devProp.maxGridSize);
}